#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <assert.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__device__ void save_blocks(int* block_offset, int range_start, int range_end, int block_size, int& block_count) {
    for (int idx = range_start; idx < range_end; idx += block_size) {
        block_offset[block_count++] = idx;
    }
}

__global__ void convert_vertical_slash_indexes_kernel(
    const int* seqlens,           // [BATCH, ]
    const int* vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    const int* slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int* block_count,             // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* block_offset,            // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_S]
    int* column_count,            // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* column_index,            // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_V]
    int N_HEADS,
    int N_ROWS,
    int BLOCK_SIZE_M,
    int BLOCK_SIZE_N,
    int NNZ_V,
    int NNZ_S
) {
    const int batch_idx = blockIdx.y;
    const int head_idx = blockIdx.x;
    const int group_idx = blockIdx.z;

    int seqlen = seqlens[batch_idx];
    int block_idx_m = group_idx * blockDim.x + threadIdx.x;
    int start_m = block_idx_m * BLOCK_SIZE_M;
    if (start_m >= seqlen) {
        return;
    }
    int end_m = start_m + BLOCK_SIZE_M;

    int row_offset = (batch_idx * N_HEADS + head_idx) * N_ROWS + block_idx_m;
    block_count += row_offset;
    block_offset += row_offset * NNZ_S;
    column_count += row_offset;
    column_index += row_offset * NNZ_V;

    int tmp_col_cnt = 0, tmp_blk_cnt = 0;
    int s = 0, v = 0;
    int v_idx = vertical_indexes[(batch_idx * N_HEADS + head_idx) * NNZ_V + v++];
    int s_idx = slash_indexes[(batch_idx * N_HEADS + head_idx) * NNZ_S + s++];

    while (s_idx >= end_m) {
        s_idx = slash_indexes[(batch_idx * N_HEADS + head_idx) * NNZ_S + s++];
    }
    s_idx = max(end_m - s_idx, BLOCK_SIZE_M);
    int range_start = s_idx - BLOCK_SIZE_M, range_end = s_idx;

    while (1) {
        if (v_idx < range_end) {
            if (v_idx < range_start) {
                column_index[tmp_col_cnt++] = v_idx;
            }
            if (v < NNZ_V) {
                v_idx = vertical_indexes[(batch_idx * N_HEADS + head_idx) * NNZ_V + v++];
            } else {
                v_idx = end_m + BLOCK_SIZE_M;
            }
        } else {
            if (s < NNZ_S) {
                s_idx = max(end_m - slash_indexes[(batch_idx * N_HEADS + head_idx) * NNZ_S + s++], BLOCK_SIZE_M);
            } else {
                save_blocks(block_offset, range_start, range_end, BLOCK_SIZE_N, tmp_blk_cnt);
                break;
            }
            if (s_idx > range_end + BLOCK_SIZE_M) {
                save_blocks(block_offset, range_start, range_end, BLOCK_SIZE_N, tmp_blk_cnt);
                range_start = s_idx - BLOCK_SIZE_M;
                range_end = s_idx;
            } else if (s_idx > range_end) {
                range_end += BLOCK_SIZE_M;
            }
        }
    }

    block_count[0] = tmp_blk_cnt;
    column_count[0] = tmp_col_cnt;
}


void convert_vertical_slash_indexes_64x64(
    const int* seqlens,           // [BATCH, ]
    const int* vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    const int* slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int* block_count,             // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* block_offset,            // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_S]
    int* column_count,            // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M)]
    int* column_index,            // [BATCH, N_HEADS, cdiv(N_CTX, BLOCK_SIZE_M), NNZ_V]
    int BATCH_SIZE,
    int N_HEADS,
    int N_ROWS,
    int NNZ_V,
    int NNZ_S
) {
    const int BLOCK_SIZE_M = 64;
    const int BLOCK_SIZE_N = 64;
    const int N_THREADS = 64;
    const dim3 dimBlock(N_THREADS);
    const dim3 dimGrid(N_HEADS, BATCH_SIZE, (N_ROWS + N_THREADS - 1) / N_THREADS);
    convert_vertical_slash_indexes_kernel<<<dimGrid, dimBlock>>>(
        seqlens, vertical_indexes, slash_indexes,
        block_count, block_offset, column_count, column_index,
        N_HEADS, N_ROWS, BLOCK_SIZE_M, BLOCK_SIZE_N, NNZ_V, NNZ_S
    );
}

std::vector<at::Tensor> convert_vertical_slash_indexes(
    torch::Tensor seqlens,           // [BATCH, ]
    torch::Tensor vertical_indexes,  // [BATCH, N_HEADS, NNZ_V]
    torch::Tensor slash_indexes,     // [BATCH, N_HEADS, NNZ_S]
    int context_size,
    int block_size_M,
    int block_size_N
) {
    assert(block_size_M == 64);
    assert(block_size_N == 64);

    hipSetDevice(seqlens.get_device());

    int batch_size = slash_indexes.size(0);
    int num_heads = slash_indexes.size(1);
    int nnz_slash = slash_indexes.size(2);
    int nnz_vertical = vertical_indexes.size(2);
    int num_rows = (context_size + block_size_M - 1) / block_size_M;

    torch::Tensor block_count = torch::zeros({batch_size, num_heads, num_rows}, seqlens.options());
    torch::Tensor block_offset = torch::zeros({batch_size, num_heads, num_rows, nnz_slash}, seqlens.options());
    torch::Tensor column_count = torch::zeros({batch_size, num_heads, num_rows}, seqlens.options());
    torch::Tensor column_index = torch::zeros({batch_size, num_heads, num_rows, nnz_vertical}, seqlens.options());

    convert_vertical_slash_indexes_64x64(
        seqlens.data_ptr<int>(),
        vertical_indexes.data_ptr<int>(),
        slash_indexes.data_ptr<int>(),
        block_count.data_ptr<int>(),
        block_offset.data_ptr<int>(),
        column_count.data_ptr<int>(),
        column_index.data_ptr<int>(),
        batch_size,
        num_heads,
        num_rows,
        nnz_vertical,
        nnz_slash
    );

    return { block_count, block_offset, column_count, column_index };
}

